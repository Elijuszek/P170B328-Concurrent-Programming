#include "hip/hip_runtime.h"
﻿#define __HIPCC__
#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>
#include <iostream>
#include <iomanip>
#include <stdio.h>
#include <fstream>

#include "nlohmann/json.hpp"

using namespace std;
using json = nlohmann::json;

class City {
public:
    char name[256];
    int population;
    double area;
    char res[256];
};

//const string dataFile = "IFF-1-1_ZekonisE_L1_dat_1.json"; // visi tinka
const string dataFile = "IFF-1-1_ZekonisE_L1_dat_2.json"; // kaikurie tinka
//const string dataFile = "IFF-1-1_ZekonisE_L1_dat_3.json"; // nei vienas netinka
const string resultFile = "IFF-1-1_ZekonisE_L1_res.txt";

void readCitiesFile(vector<City>* cities) {
    ifstream stream(dataFile);
    json allCitiesJson = json::parse(stream);

    auto allItems = allCitiesJson["cities"];
    for (auto& new_items : allItems) {
        City tempItem;
        string n = new_items["name"];
        //Returns a pointer to an array that contains a null-terminated sequence of characters (i.e., a C-string) 
        // representing the current value of the string object.
        strcpy(tempItem.name, n.c_str());
        tempItem.population = new_items["population"];
        tempItem.area = new_items["area"];
        cities->push_back(tempItem);
    }
    stream.close();
}

void writeListToFile(vector<City>& cities, string fileName) {
    ofstream file;
    file.open(fileName, ios::out);
    file << setw(33) << "Pradiniai duomenys" << endl
        << "--------------------------------------------------------------" << endl
        << setw(5) << "Nr. |" << setw(30) << "Name |" << setw(15) << "Population |" << setw(17) << "Area |" << endl
        << "--------------------------------------------------------------" << endl;

    for (int i = 0; i < cities.size(); i++)
    {
        file << setw(5) << to_string(i+1) << setw(30) << cities[i].name << " |" << setw(13) << to_string(cities[i].population) << " |"
            << setw(15) << to_string(cities[i].area) << " |" << endl;
    }
    file << "--------------------------------------------------------------" << endl << endl;
    file.close();
}

void writeResultToFile(City cities[], string fileName, int res_size) {
    ofstream file;
    file.open(fileName, ios::app);
    file << setw(39) << "Rezultatai" << endl
        << "---------------------------------------------------------------------------------------" << endl
        << setw(5) << "Nr. |" << setw(30) << "Name |" << setw(15) << "Population |" << setw(17) << "Area |" << setw(19) << "Teksto rezultatas   |" << endl
        << "---------------------------------------------------------------------------------------" << endl;

    for (int i = 0; i < res_size; i++)
    {
        file << setw(5) << to_string(i+1) << setw(30) << cities[i].name << " |" << setw(13) << to_string(cities[i].population) << " |"
            << setw(15) << to_string(cities[i].area) << " |" << setw(17) << (cities[i].res) << " |" << endl;
    }
    file << "---------------------------------------------------------------------------------------" << endl;
    file.close();
}


__device__ void gpu_strcpy(char* dest, const char* src) {
	int i = 0;
	do {
		dest[i] = src[i];
	} while (src[i++] != 0);
}

__device__ void gpu_string(char* dest, const char* src) {
	dest[0] = src[0];
	dest[1] = src[1];
	dest[2] = '<';
	dest[3] = '1';
	dest[4] = '0';
	dest[5] = '0';
}

__global__ void gpu_func(City* device_cities, City* device_results, int* device_array_size, int* device_slice_size, int* device_result_count) {
    // Compute start index
    unsigned long start_index =* device_slice_size * threadIdx.x;
    unsigned long end_index;

    if (threadIdx.x == blockDim.x - 1) {
        end_index = *device_array_size;
    }
    else {
        end_index = *device_slice_size * (threadIdx.x + 1);
    }

    auto fp_sum = 0;

    for (int i = start_index; i < end_index; i++) {
        double population = device_cities[i].population;
        double area = device_cities[i].area;
        double density = population / area;
        if (density > 100) {
            City city;
            gpu_strcpy(city.name, device_cities[i].name);
            city.population = device_cities[i].population;
            city.area = device_cities[i].area;
            gpu_string(city.res, device_cities[i].name);

            // Inserting into results array
            int index = atomicAdd(device_result_count, 1);
            device_results[index] = city;
		}
	}
}

const int SIZE = 256;
const int BLOCKS = 2;
const int THREADS = 4;

int main()
{
	vector<City> data;
    readCitiesFile(&data);
    City* cities = &data[0];
    City results[SIZE];

    int slice_size = SIZE / THREADS;
    int result_count = 0;

    City* device_cities;
    City* device_results;
    int* device_array_size;
    int* device_slice_size;
    int* device_result_count;


    // Allocate memory on GPU
    hipMalloc((void**)&device_cities, SIZE * sizeof(City));
    hipMalloc((void**)&device_results, SIZE * sizeof(City));
    hipMalloc((void**)&device_array_size, sizeof(int));
    hipMalloc((void**)&device_slice_size, sizeof(int));
    hipMalloc((void**)&device_result_count, sizeof(int));

    //Funkcijos, vykdomos GPU ir kviečiamos iš GPU
    //hipMemcpyHostToHost iš CPU į CPU
    //hipMemcpyHostToDevice iš CPU į GPU
    //hipMemcpyDeviceToHost iš GPU į CPU
    //hipMemcpyDeviceToDevice iš GPU į GPU

    // Copy data from CPU to GPU
    hipMemcpy(device_cities, cities, SIZE * sizeof(City), hipMemcpyHostToDevice);
    hipMemcpy(device_array_size, &SIZE, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(device_slice_size, &slice_size, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(device_result_count, &result_count, sizeof(int), hipMemcpyHostToDevice);

    gpu_func<<<BLOCKS, THREADS>>>(device_cities, device_results, device_array_size, device_slice_size, device_result_count);

    // Blocks CPU code until GPU code is done
    hipDeviceSynchronize();

    hipMemcpy(&results, device_results, SIZE * sizeof(City), hipMemcpyDeviceToHost);
    int RES_SIZE = 0;
    hipMemcpy(&RES_SIZE, device_result_count, sizeof(int), hipMemcpyDeviceToHost);

    writeListToFile(data, resultFile);
    
    writeResultToFile(results, resultFile, RES_SIZE);

    // Free memory on 
    hipFree(device_cities);
    hipFree(device_results);
    hipFree(device_array_size);
    hipFree(device_slice_size);
    hipFree(device_result_count);

    return 0;
}